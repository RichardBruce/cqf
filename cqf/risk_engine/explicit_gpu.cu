
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "hip/hip_runtime_api.h"

/* Scratch space positions */
const int max_ns    = 1024;

const int lower_delta_pos       = 0;
const int mid_delta_pos         = lower_delta_pos + max_ns;
const int upper_delta_pos       = mid_delta_pos + max_ns;

const int lower_gamma_pos       = upper_delta_pos + max_ns;
const int mid_gamma_pos         = lower_gamma_pos + max_ns;
const int upper_gamma_pos       = mid_gamma_pos + max_ns;

const int matrix_equal_pos      = upper_gamma_pos + max_ns;

const int scratch_space_size    = matrix_equal_pos + max_ns;


void print_cuda_error(hipError_t err, char *at)
{
    if (err)
    {
        printf("Error from CUDA at : %s\n", at);
        printf("Message: %s\n", hipGetErrorString(err));
    }
}


__device__ float call_payoff(const float s, const float k)
{
    return fmaxf(0.0f, s - k);
}


__device__ void get_coeffs(const float *const grid, float *const scratch, const int ns, const int i)
{
    /* Difference vs. the grid below */
    float d0;
    float d1;
    if (i == 0)
    {
        d0 = grid[1] - grid[0];
        d1 = grid[2] - grid[1];
    }
    else if (i == (ns - 1))
    {
        d0 = grid[i - 1] - grid[i - 2];
        d1 = grid[i] - grid[i - 1];
    }
    else
    {
        d0 = grid[i] - grid[i - 1];
        d1 = grid[i + 1] - grid[i];
    }
    const float d1_p_d2 = d0 + d1;

    /* Delta coeffs */
    /* Middle */
    if ((i != 0) & (i != (ns - 1)))
    {
        scratch[lower_delta_pos + i] = -d1 / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = (d1 - d0) / (d0 * d1);
        scratch[upper_delta_pos + i] = d0 / (d1 * d1_p_d2);
    }
    /* Lower boundary */
    else if (i == 0)
    {
        scratch[lower_delta_pos + i] = (-2.0f * d0 - d1) / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = d1_p_d2 / (d0 * d1);
        scratch[upper_delta_pos + i] = -d0 / (d1 * d1_p_d2);
    }
    /* Upper boundary */
    else if (i == (ns - 1))
    {
        scratch[lower_delta_pos + i] = d1 / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = (-d0 - d1) / (d0 * d1);
        scratch[upper_delta_pos + i] = (d0 + 2.0f * d1) / (d1 * d1_p_d2);
    }

    /* Gamma coeffs */
    /* Middle */
    if ((i != 0) & (i != (ns - 1)))
    {
        scratch[lower_gamma_pos + i]  =  2.0f / (d0 * d1_p_d2);
        scratch[mid_gamma_pos   + i]  = -2.0f / (d0 * d1);
        scratch[upper_gamma_pos + i]  =  2.0f / (d1 * d1_p_d2);
    }
    __syncthreads();
}


/* Populate the matrix */
__device__ void explicit_step(float *const scratch, float *const matrix_equal, const float *const tp1, const float *const grid, 
    const float half_sigma_sq, const float r, const float t_inc, const int ns, const int i)
{
    /* Boundary conditions */
    /* s = 0.0 */
    if (i == 0)
    {
        const float b = -r * t_inc;
        matrix_equal[0] = (1.0f + b) * tp1[0];
    }
    /* s = s_max*/
    else if (i == (ns - 1))
    {
        const float r_s = r * grid[ns - 1];

        const float a = -r_s * t_inc;
        const float b = -(r - r_s) * t_inc;
        matrix_equal[ns - 1]  = a * tp1[ns - 2];
        matrix_equal[ns - 1] += (1.0f + b) * tp1[ns - 1];
    }
    else if (i < ns)
    {
        const float g = half_sigma_sq * grid[i] * grid[i];
        const float r_s = r * grid[i];
            
        const float a = ((scratch[lower_delta_pos + i] * r_s) + (scratch[lower_gamma_pos + i] * g))     * t_inc;
        const float b = ((scratch[mid_delta_pos + i]   * r_s) + (scratch[mid_gamma_pos + i]   * g) - r) * t_inc;
        const float c = ((scratch[upper_delta_pos + i] * r_s) + (scratch[upper_gamma_pos + i] * g))     * t_inc;
        matrix_equal[i]  = a * tp1[i - 1];
        matrix_equal[i] += (1.0f + b) * tp1[i];
        matrix_equal[i] += c * tp1[i + 1];
    }

    __syncthreads();
}


__global__ void explicit_scheme(const float *const grid, float *const scratch, const float half_sigma_sq, const float r, 
    const float t_inc, const float k, const int ns, const int nt)
{
    const int i = threadIdx.x;
    if (ns & 0x1f)
    {
        /* Only multiple of 32 space steps are supported */
        return;
    }
    
    /* Move grid to shared memory, needed for off by 1 access and reused */
    __shared__ float shared_equal[max_ns];
    shared_equal[i] = grid[i];
    __syncthreads();

    /* Build grid based coeffs, completely parrallel */
    __shared__ float shared_tp1[max_ns];
    shared_tp1[i] = call_payoff(shared_equal[i], k);
    get_coeffs(shared_equal, scratch, ns, i);
    
    /* Solve back in time */
    for (unsigned int j = 0; j < nt >> 1; ++j)
    {
        explicit_step(scratch, shared_equal, shared_tp1, grid, half_sigma_sq, r, t_inc, ns, i);
        shared_equal[i] = fmaxf(shared_equal[i], call_payoff(shared_equal[i], k));

        explicit_step(scratch, shared_tp1, shared_equal, grid, half_sigma_sq, r, t_inc, ns, i);
        shared_tp1[i] = fmaxf(shared_tp1[i], call_payoff(shared_tp1[i], k));
    }

    scratch[matrix_equal_pos + i] = shared_tp1[i];
}


void american_call_test()
{
    /* Pricing set up */
    printf("Pricing American Call\n");
    const unsigned int ns = 1024;   /* Want multiples of warp size (32) */

    const float s = 100.0f;
    const float r = 0.05f;
    const float sigma = 0.2f;
    const float half_sigma_sq = 0.5f * sigma * sigma;

    const float k = 100.0f;
    const float t = 1.0f;
    const float t_inc   = 0.9f / (static_cast<float>(ns * ns) * sigma * sigma);
    const int nt        = static_cast<int>(t / t_inc) + 1;

    /* Build regular grid based at 0 */
    float *grid = new float [ns];
    const float s_inc = (s * 3.0f) / ns;
    for (unsigned int i = 0; i < ns; ++i)
    {
        grid[i] = i * s_inc;
    }

    print_cuda_error(hipSetDevice(0), "Set device");

    /* Prepare device memory */
    float *dev_grid;
    print_cuda_error(hipMalloc((void **)&dev_grid, ns * sizeof(float)), "Malloc grid");
    print_cuda_error(hipMemcpy(dev_grid, grid, ns * sizeof(float), hipMemcpyHostToDevice), "Copy grid to device");

    float *dev_scratch;
    print_cuda_error(hipMalloc((void **)&dev_scratch, scratch_space_size * sizeof(float)), "Malloc scratch");
    print_cuda_error(hipMemset(dev_scratch, 0, scratch_space_size * sizeof(float)), "Clear scratch");

    /* Call kernels */
    hipProfilerStart();
    explicit_scheme<<<1, ns>>>(dev_grid, dev_scratch, half_sigma_sq, r, t_inc, k, ns, nt);
    hipProfilerStop();
    print_cuda_error(hipGetLastError(), "Kernel execution");

    float *res = new float [ns];
    print_cuda_error(hipMemcpy(res, &dev_scratch[matrix_equal_pos], ns * sizeof(float), hipMemcpyDeviceToHost), "Copy grid to host");
    for (unsigned int i = 0; i < ns; ++i)
    {
        printf("%.2f: %.2f\n", grid[i], res[i]);
    }

    /* Clean up */
    print_cuda_error(hipFree(dev_grid), "Free grid");
    print_cuda_error(hipFree(dev_scratch), "Free scratch");

    print_cuda_error(hipDeviceReset(), "Device reset");

    delete [] grid;
    delete [] res;
}


int main()
{
    american_call_test();

    return 0;
}
