
#include <hip/hip_runtime.h>
#include <stdio.h>

const int max_threads_per_block = 512;

#define HANDLE_CUDA_ERROR(err)	if (err) { printf("%s", hipGetErrorString(err)); return; }



__device__ int cyclic_reduction_forward_reduction(float *lower, float *diagonal, float *upper, float *equal, const int dim, int step, const int to)
{
	/* Forward reduction */
	for (; (step * to * 3) <= dim; step <<= 1)
	{
		const int addr = (threadIdx.x * (step << 1)) + (step << 1) - 1;
		if (addr < dim)
		{
			if (addr - step >= 0)
			{
				const float alpha = -lower[addr] / diagonal[addr - step];
				equal[addr]    += (alpha * equal[addr - step]);
				diagonal[addr] += (alpha * upper[addr - step]);
				lower[addr]		= alpha * lower[addr - step];
			}

			if (addr + step < dim)
			{
				const float gamma = -upper[addr] / diagonal[addr + step];
				equal[addr]	   += (gamma * equal[addr + step]);
				diagonal[addr] += (gamma * lower[addr + step]);
				upper[addr]		= gamma * upper[addr + step];
			}
		}
		__syncthreads();
	}

	return step;
}


__device__ void cyclic_reduction_back_substitution(float *lower, float *diagonal, float *upper, float *equal, const int dim, int step, const int to)
{
	/* Backward substitution */
	for (; step > to; step >>= 1)
	{
		const int addr = (threadIdx.x * (step << 1)) + step - 1;
		if (addr < dim)
		{
			if (addr - step >= 0)
			{
				equal[addr] -= (lower[addr] * equal[addr - step]);
			}

			if (addr + step < dim)
			{
				equal[addr] -= (upper[addr] * equal[addr + step]);
			}

			equal[addr] = equal[addr] / diagonal[addr];
		}
		__syncthreads();
	}
}


__global__ void cyclic_reduction_device(float *lower_glb, float *diagonal_glb, float *upper_glb, float *equal_glb, const int dim)
{
	__shared__ float lower[512];
	__shared__ float diagonal[512];
	__shared__ float upper[512];
	__shared__ float equal[512];

	lower[threadIdx.x] = lower_glb[threadIdx.x];
	diagonal[threadIdx.x] = diagonal_glb[threadIdx.x];
	upper[threadIdx.x] = upper_glb[threadIdx.x];
	equal[threadIdx.x] = equal_glb[threadIdx.x];
	__syncthreads();

	/* Forward reduction */
	int step = cyclic_reduction_forward_reduction(lower, diagonal, upper, equal, dim, 1, 1);

	/* Solve base system */
	if (threadIdx.x == 0)
	{
		if ((dim / step) == 2) /* Solve simultaneous equations */
		{
			const int equal_addr = (step << 1) - 1;
			const float a0 = diagonal[equal_addr - step];
			const float a1 = lower[equal_addr];
			const float b0 = upper[equal_addr - step];
			const float b1 = diagonal[equal_addr];
			const float c0 = equal[equal_addr - step];
			const float c1 = equal[equal_addr];

			equal[equal_addr] = (c0 * a1 - a0 * c1) / (a1 * b0 - a0 * b1);
			equal[equal_addr - step] = (c0 - b0 * equal[equal_addr]) / a0;
		}
		else /* blk_size == 1, equations are already solved */
		{
			const int equal_addr = step - 1;
			equal[equal_addr] = equal[equal_addr] / diagonal[equal_addr];
		}
	}

	__syncthreads();
	step >>= 1;

	/* Backward substitution */
	cyclic_reduction_back_substitution(lower, diagonal, upper, equal, dim, step, 0);
	equal_glb[threadIdx.x] = equal[threadIdx.x];
}


void cyclic_reduction(float *lower, float *diagonal, float *upper, float *equal, const int dim)
{
	const int log_dim = static_cast<int>(ceil(log(static_cast<float>(dim)) / log(2.0f)));

	/* Get device */
	HANDLE_CUDA_ERROR(hipSetDevice(0));

	/* Allocate and copy memory */
	float *dev_equal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_equal, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_equal, equal, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_lower;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_lower, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_lower, lower, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_diagonal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_diagonal, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_diagonal, diagonal, dim * sizeof(float), hipMemcpyHostToDevice));
	
	float *dev_upper;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_upper, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_upper, upper, dim * sizeof(float), hipMemcpyHostToDevice));

	/* Run kernel */
	if (dim > max_threads_per_block)
	{
		printf("Thead count (%i) exceeds maximum", dim);
		return;
	}
	cyclic_reduction_device<<<1, dim>>>(dev_lower, dev_diagonal, dev_upper, dev_equal, dim);

	/* Copy data back */
	HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	HANDLE_CUDA_ERROR(hipMemcpy(equal, dev_equal, dim * sizeof(float), hipMemcpyDeviceToHost));

	/* Clean up */
	HANDLE_CUDA_ERROR(hipFree(dev_equal));
	HANDLE_CUDA_ERROR(hipFree(dev_lower));
	HANDLE_CUDA_ERROR(hipFree(dev_diagonal));
	HANDLE_CUDA_ERROR(hipFree(dev_upper));

	/* Flush profiling info */
	HANDLE_CUDA_ERROR(hipDeviceReset());
}


__global__ void parallel_cyclic_reduction_device(float *lower_glb, float *diagonal_glb, float *upper_glb, float *equal_glb, const int dim)
{
	const int rank = threadIdx.x;
	__shared__ float lower[512];
	__shared__ float diagonal[512];
	__shared__ float upper[512];
	__shared__ float equal[512];

	lower[rank] = lower_glb[rank];
	diagonal[rank] = diagonal_glb[rank];
	upper[rank] = upper_glb[rank];
	equal[rank] = equal_glb[rank];
	__syncthreads();

	float lower_tmp;
	float upper_tmp;
	float result_tmp;
	float diag_tmp;
	for (int span = 1 ; span < dim; span <<= 1)
	{
		if (rank < dim)
		{
			result_tmp = equal[rank];
			diag_tmp = diagonal[rank];

			if (rank - span >= 0)
			{
				lower_tmp = -lower[rank] / diagonal[rank - span];
				diag_tmp += lower_tmp * upper[rank - span];
				result_tmp += lower_tmp * equal[rank - span];
				lower_tmp *= lower[rank - span];
			}

			if (rank + span < dim)
			{
				upper_tmp = -upper[rank] / diagonal[rank + span];
				diag_tmp += upper_tmp * lower[rank + span];
				result_tmp += upper_tmp * equal[rank + span];
				upper_tmp *= upper[rank + span];
			}
		}
		__syncthreads();

		if (rank < dim)
		{
			lower[rank] = lower_tmp;
			upper[rank] = upper_tmp;
			equal[rank] = result_tmp;
			diagonal[rank] = diag_tmp;
		}
		__syncthreads();
	}

	if (rank < dim)
	{
		equal_glb[rank] = equal[rank] / diagonal[rank];
	}
}


void parallel_cyclic_reduction(float *lower, float *diagonal, float *upper, float *equal, const int dim)
{
	/* Get device */
	HANDLE_CUDA_ERROR(hipSetDevice(0));

	/* Allocate and copy memory */
	float *dev_equal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_equal, dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_equal, equal, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_lower;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_lower, dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_lower, lower, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_diagonal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_diagonal, dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_diagonal, diagonal, dim * sizeof(float), hipMemcpyHostToDevice));
	
	float *dev_upper;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_upper, dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_upper, upper, dim * sizeof(float), hipMemcpyHostToDevice));

	/* Run kernel */
	if (dim > max_threads_per_block)
	{
		printf("Thead count (%i) exceeds maximum", dim);
		return;
	}
	parallel_cyclic_reduction_device<<<1, dim>>>(dev_lower, dev_diagonal, dev_upper, dev_equal, dim);

	/* Copy data back */
	HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	HANDLE_CUDA_ERROR(hipMemcpy(equal, dev_equal, dim * sizeof(float), hipMemcpyDeviceToHost));

	/* Clean up */
	HANDLE_CUDA_ERROR(hipFree(dev_equal));
	HANDLE_CUDA_ERROR(hipFree(dev_lower));
	HANDLE_CUDA_ERROR(hipFree(dev_diagonal));
	HANDLE_CUDA_ERROR(hipFree(dev_upper));

	/* Flush profiling info */
	HANDLE_CUDA_ERROR(hipDeviceReset());
}


__global__ void hybrid_cyclic_reduction_device(float *lower_glb, float *diagonal_glb, float *upper_glb, float *equal_glb, const int dim)
{
	__shared__ float lower[512];
	__shared__ float diagonal[512];
	__shared__ float upper[512];
	__shared__ float equal[512];

	lower[threadIdx.x] = lower_glb[threadIdx.x];
	diagonal[threadIdx.x] = diagonal_glb[threadIdx.x];
	upper[threadIdx.x] = upper_glb[threadIdx.x];
	equal[threadIdx.x] = equal_glb[threadIdx.x];
	__syncthreads();


	/* Cyclic forward reduction */
	int step = cyclic_reduction_forward_reduction(lower, diagonal, upper, equal, dim, 1, 128);

	/* Parallel cyclic reduction to solve system */
	float lower_tmp;
	float upper_tmp;
	float result_tmp;
	float diag_tmp;
	const int rank = (threadIdx.x * step) + step - 1;
	for (int span = step; span < dim; span <<= 1)
	{
		if (rank < dim)
		{
			result_tmp = equal[rank];
			diag_tmp = diagonal[rank];

			if (rank - span >= 0)
			{
				lower_tmp = -lower[rank] / diagonal[rank - span];
				diag_tmp += lower_tmp * upper[rank - span];
				result_tmp += lower_tmp * equal[rank - span];
				lower_tmp *= lower[rank - span];
			}

			if (rank + span < dim)
			{
				upper_tmp = -upper[rank] / diagonal[rank + span];
				diag_tmp += upper_tmp * lower[rank + span];
				result_tmp += upper_tmp * equal[rank + span];
				upper_tmp *= upper[rank + span];
			}
		}
		__syncthreads();

		if (rank < dim)
		{
			lower[rank] = lower_tmp;
			upper[rank] = upper_tmp;
			equal[rank] = result_tmp;
			diagonal[rank] = diag_tmp;
		}
		__syncthreads();
	}

	if (rank < dim)
	{
		equal[rank] /= diagonal[rank];
	}
	__syncthreads();


	/* Cyclic backward substitution */
	cyclic_reduction_back_substitution(lower, diagonal, upper, equal, dim, step >> 1, 0);
	equal_glb[threadIdx.x] = equal[threadIdx.x];
}


void hybrid_cyclic_reduction(float *lower, float *diagonal, float *upper, float *equal, const int dim)
{
	const int log_dim = static_cast<int>(ceil(log(static_cast<float>(dim)) / log(2.0f)));

	/* Get device */
	HANDLE_CUDA_ERROR(hipSetDevice(0));

	/* Allocate and copy memory */
	float *dev_equal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_equal, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_equal, equal, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_lower;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_lower, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_lower, lower, dim * sizeof(float), hipMemcpyHostToDevice));

	float *dev_diagonal;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_diagonal, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_diagonal, diagonal, dim * sizeof(float), hipMemcpyHostToDevice));
	
	float *dev_upper;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_upper, dim * log_dim * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_upper, upper, dim * sizeof(float), hipMemcpyHostToDevice));

	/* Run kernel */
	if (dim > max_threads_per_block)
	{
		printf("Thead count (%i) exceeds maximum", dim);
		return;
	}
	hybrid_cyclic_reduction_device<<<1, dim>>>(dev_lower, dev_diagonal, dev_upper, dev_equal, dim);

	/* Copy data back */
	HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	HANDLE_CUDA_ERROR(hipMemcpy(equal, dev_equal, dim * sizeof(float), hipMemcpyDeviceToHost));

	/* Clean up */
	HANDLE_CUDA_ERROR(hipFree(dev_equal));
	HANDLE_CUDA_ERROR(hipFree(dev_lower));
	HANDLE_CUDA_ERROR(hipFree(dev_diagonal));
	HANDLE_CUDA_ERROR(hipFree(dev_upper));

	/* Flush profiling info */
	HANDLE_CUDA_ERROR(hipDeviceReset());
}
