#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <windows.h>



void serialFir(float *filtered, const float *values, const float *coeffs, unsigned int size, unsigned int taps)
{
	const time_t start_time = time(0);
	fprintf(stdout, "Serial start time: %d\n", start_time);

	// For each value
	for (unsigned int i = 0; i < size - taps; ++i)
	{
		// MAC accross tap coeffs
		float res = 0.f;
		for (unsigned int j = 0; j < taps; ++j)
		{
			const float v = values[i + j];
			const float c = coeffs[j];
			res += v * c;
		}
		filtered[i] = res;
	}

	fprintf(stdout, "Serial end time: %d\n", time(0));
	fprintf(stdout, "Serial run time: %d\n", time(0) - start_time);
}


// Parallel fir kernel
__global__ void firKernel(float *filtered, const float *values, const float *coeffs, const int size, const int taps)
{
	// Copy the coeffs
	extern __shared__ float s_coeffs[];
	const int coeff_blocks = (int)((taps / (float)blockDim.x) + 0.5f);
	for (unsigned int i = 0; i < (coeff_blocks * blockDim.x); i += blockDim.x)
	{
		if (i < taps)
		{
			s_coeffs[i] = coeffs[i];
		}
	}

	// Wait for all threads
	__syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = threadIdx.x;

	// MAC accross tap coeffs
	float res = 0.f;
	for (unsigned int j = 0; j < taps; ++j)
	{
		res += values[i + j] * s_coeffs[j];
	}
	//filtered[(blockIdx.x * blockDim.x) + i] = res;
	filtered[i] = res;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t firWithCuda(float *filtered, const float *values, const float *coeffs, unsigned int size, unsigned int taps)
{
    float *dev_values = 0;
    float *dev_coeffs = 0;
    float *dev_filtered = 0;
    hipError_t cudaStatus;
	
    const int threadCount = 64;
    const int blockCount = ((size - taps) / threadCount) + ((size - taps) % threadCount == 0?0:1); 


	const time_t start_time = time(0);
	fprintf(stdout, "Parallel start time: %d\n", start_time);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed: %s", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_values, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_coeffs, taps * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_filtered, (size - taps) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_values, values, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_coeffs, coeffs, taps * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    firKernel<<<blockCount, threadCount, taps * sizeof(float)>>>(dev_filtered, dev_values, dev_coeffs, size, taps);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error: %s!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(filtered, dev_filtered, (size - taps) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	fprintf(stdout, "Parallel end time: %d\n", time(0));
	fprintf(stdout, "Parallel run time: %d\n", time(0) - start_time);

Error:
    hipFree(dev_values);
    hipFree(dev_coeffs);
    hipFree(dev_filtered);
    
    return cudaStatus;
}


/*int main()
{
	const float EPSILON = 1.0e-3f;

    const int size = 1048576;//131072;//4194304;
	const int taps = 1024;//8192;//8192;
    float *values;
    float *coeffs;
    float *serial_filtered;
	float *parallel_filtered;

	values = (float *)malloc(size * sizeof(float));
	coeffs = (float *)malloc(taps * sizeof(float));
	serial_filtered = (float *)malloc((size - taps) * sizeof(float));
	parallel_filtered = (float *)malloc((size - taps) * sizeof(float));

	// Initialise data
	for (unsigned int i = 0; i < size; ++i)
	{
		values[i] = (float)sin(2. * M_PI * (i / (double)size));
	}

	for (unsigned int i = 0; i < taps; ++i)
	{
		coeffs[i] = (float)cos(2. * M_PI * (i / (double)taps));
	}

	// Run serial kernel
	serialFir(serial_filtered, values, coeffs, size, taps);

    // Run parallel fir
    hipError_t cudaStatus = firWithCuda(parallel_filtered, values, coeffs, size, taps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "firWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	// Check output
	for (unsigned int i = 0; i < size - taps; ++i)
	{
		if (fabs(serial_filtered[i] - parallel_filtered[i]) > EPSILON)
		{
			fprintf(stdout, "Mismatch at %i, expected: %f, got %f\n", i, serial_filtered[i], parallel_filtered[i]);
		}
	}

	// Clean up
	free(values);
	free(coeffs);
	free(serial_filtered);
	free(parallel_filtered);

    return 0;
}*/
