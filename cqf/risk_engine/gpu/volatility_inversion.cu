#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "hip/hip_runtime_api.h"

#include "volatility_inversion.cuh"


#ifndef PI 
#define PI 3.141592653589793238462643f
#endif 

const int serial_threads_per_block = 448;

const int parallel_guesses_per_option = 4;
const int parallel_threads_per_block = 192;

const int vega_parallel_guesses_per_option = 4;
const int vega_parallel_threads_per_block = 192;


#define HANDLE_CUDA_ERROR(err)	if (err) { printf("%s", hipGetErrorString(err)); return; }

__device__ __host__ float normal_pdf(const float z) 
{
	return (1.0f / sqrt(2.0f * PI)) * exp(-0.5f * z);
}


__device__ __host__ float normal_cdf(const float z)
{
    const float b1 =  0.31938153f; 
    const float b2 = -0.356563782f; 
    const float b3 =  1.781477937f;
    const float b4 = -1.821255978f;
    const float b5 =  1.330274429f; 
    const float p  =  0.2316419f; 
    const float c2 =  0.3989423f; 

    if (z >  6.0f)
	{
		return 1.0f;
	}

    if (z < -6.0f)
	{
		return 0.0f;
	}

    const float a = abs(z); 
    const float t = 1.0f / (1.0f + a * p); 
    const float b = c2 * exp((-z) * (z / 2.0f)); 
    
	float n = ((((b5 * t + b4) * t + b3) * t + b2) * t + b1) * t; 
    n = 1.0f - b * n; 
    if ( z < 0.0f )
	{
		n = 1.0f - n;
	}
    return n; 
}


__device__ __host__ float call_price(const float s, const float r, const float v, const float t, const float k)
{
	const float sqrt_t = sqrt(t);
	const float d1 = (1.0f / (v * sqrt_t)) * (log(s / k) + (r + v * v * 0.5f) * t);
	const float d2 = d1 - v * sqrt_t;

	return (s * normal_cdf(d1)) - (k * exp(-r * t) * normal_cdf(d2));
}


__device__ __host__ float call_vega(const float s, const float r, const float v, const float t, const float k)
{
	const float sqrt_t = sqrt(t);
	const float d1 = (1.0f / (v * sqrt_t)) * (log(s / k) + (r + v * v * 0.5f) * t);
	return s * normal_pdf(d1) * sqrt_t;
}


__device__ __host__ float put_price(const float s, const float r, const float v, const float t, const float k)
{
	const float sqrt_t = sqrt(t);
	const float d1 = (1.0f / (v * sqrt_t)) * (log(s / k) + (r + v * v * 0.5f) * t);
	const float d2 = d1 - v * sqrt_t;

	return (k * exp(-r * t) * normal_cdf(-d2)) - (s * normal_cdf(-d1));
}


__device__ __host__ float put_vega(const float s, const float r, const float v, const float t, const float k)
{
	return call_vega(s, r, v, t, k);
}


__global__ void volatility_inversion_device(const float s, const float r, float *v, const float *t, const float *k, 
	const float *p, const float tol, const int iter, const int num)
{
	const int option = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (option >= num)
	{
		return;
	}

	float v_local = v[option];
	float error;
	int i = 0;
	do
	{
		const float price = call_price(s, r, v_local, t[option], k[option]);
		const float vega = call_vega(s, r, v_local, t[option], k[option]);
		
		error = (p[option] - price);
		v_local += error / vega;
	} while ((abs(error) > tol) && (i++ < iter));

	v[option] = v_local;
}

template<int GUESSES, int BLOCK_SIZE>
__device__ int find_best_guess(const float *const ferr_ladder, const int tid, const int lower_guess)
{
	__shared__ int smallest_idx[BLOCK_SIZE];
	__shared__ float smallest_error[BLOCK_SIZE];

	/* Find minimum error */
	smallest_idx[tid] = tid;
	smallest_error[tid] = ferr_ladder[tid];
	if ((GUESSES > 16) && (smallest_error[tid + 16] < smallest_error[tid]))
	{
		smallest_error[tid] = smallest_error[tid + 16];
		smallest_idx[tid] = tid + 16;
	}

	if ((GUESSES > 8) && (smallest_error[tid + 8] < smallest_error[tid]))
	{
		smallest_error[tid] = smallest_error[tid + 8];
		smallest_idx[tid] = tid + 8;
	}

	if ((GUESSES > 4) && (smallest_error[tid + 4] < smallest_error[tid]))
	{
		smallest_error[tid] = smallest_error[tid + 4];
		smallest_idx[tid] = tid + 4;
	}

	if ((GUESSES > 2) && (smallest_error[tid + 2] < smallest_error[tid]))
	{
		smallest_error[tid] = smallest_error[tid + 2];
		smallest_idx[tid] = tid + 2;
	}

	if (smallest_error[tid + 1] < smallest_error[tid])
	{
		smallest_error[tid] = smallest_error[tid + 1];
		smallest_idx[tid] = tid + 1;
	}
	
	return smallest_idx[lower_guess];
}


template<int GUESSES>
__global__ void parallel_volatility_inversion_device(const float s, const float r, float *v, const float *t, const float *k, 
	const float *p, const float tol, const int iter)
{
	const int tid = threadIdx.x;
	const int guess = tid & (GUESSES - 1);
	const int lower_guess = tid & ~(GUESSES - 1);
	const int upper_guess = lower_guess + GUESSES - 1;
	const int log_guesses = 31 - __clz(GUESSES);
	const int option = (blockIdx.x * (parallel_threads_per_block >> log_guesses)) + (tid >> log_guesses);

	/* Build guesses */
	__shared__ float v_ladder[parallel_threads_per_block];
	__shared__ float err_ladder[parallel_threads_per_block];
	__shared__ float ferr_ladder[parallel_threads_per_block];

	float ladder_span = 0.04f;	/* Span the guesses over 4% */
	float v_min = v[option] - (ladder_span * 0.5f);

	int i = 0;
	const float guess_fraction = guess / static_cast<float>(GUESSES - 1);
	do
	{
		/* Work out guess */
		v_ladder[tid] = v_min + (ladder_span * guess_fraction);

		/* Price */
		err_ladder[tid] = call_price(s, r, v_ladder[tid], t[option], k[option]) - p[option];
		ferr_ladder[tid] = abs(err_ladder[tid]);
		
		/* Find minimum error */
		const int min_err_pos = find_best_guess<GUESSES, parallel_threads_per_block>(ferr_ladder, tid, lower_guess);
		if (ferr_ladder[min_err_pos] < tol)
		{
			if (tid == lower_guess)
			{
				v[option] = v_ladder[min_err_pos];
			}
			break;
		}

		/* Pick the span for the next ladder */
		/* Doesnt matter if v_min is actually higher than v_max so long as 0 is crossed */
		if ((err_ladder[lower_guess] * err_ladder[upper_guess]) >= 0.0f) /* Root not found (or two roots found) */
		{
			ladder_span *= 2.0f;
			if (abs(err_ladder[lower_guess] - err_ladder[upper_guess]) < tol) /* Ladder is very flat so no direction */
			{
				v_min -= ladder_span * 0.5f;
				ladder_span *= 2.0f;
			}
			else if (ferr_ladder[lower_guess] < ferr_ladder[upper_guess]) /* Lower end is closer to root so expand it */
			{
				v_min -= ladder_span;
			}
			else /* Upper end is closer to root so expand it */
			{
				v_min = v_ladder[upper_guess];
			}
		}
		else if (min_err_pos == lower_guess) /* Root found at lower extreme of ladder */
		{
			if ((err_ladder[lower_guess] * err_ladder[lower_guess + 1]) >= 0.0f)
			{
				ladder_span *= 2.0f;
				v_min = (v_ladder[min_err_pos] - ladder_span);
			}
			else
			{
				ladder_span *= 0.5f;
				v_min = v_ladder[min_err_pos];
			}
		}
		else if (min_err_pos == upper_guess) /* Root found at upper extreme of ladder */
		{
			if ((err_ladder[upper_guess] * err_ladder[upper_guess - 1]) >= 0.0f)
			{
				ladder_span *= 2.0f;
				v_min = v_ladder[min_err_pos];
			}
			else
			{
				ladder_span *= 0.5f;
				v_min = v_ladder[upper_guess - 1];
			}
		}
		else if ((err_ladder[min_err_pos] * err_ladder[min_err_pos - 1]) < 0.0f) /* Root in bin below min error */
		{
			ladder_span *= 1.0f / GUESSES;
			v_min = v_ladder[min_err_pos - 1];
		}
		else /* Root in bin above min error */
		{
			ladder_span *= 1.0f / GUESSES;
			v_min = v_ladder[min_err_pos];
		}
	} while (i++ < iter);
}


template<int GUESSES>
__global__ void vega_guided_parallel_volatility_inversion_device(const float s, const float r, float *v, const float *t, const float *k, 
	const float *p, const float tol, const int iter)
{
	const int tid = threadIdx.x;
	const int guess = tid & (GUESSES - 1);
	const int lower_guess = tid & ~(GUESSES - 1);
	const int upper_guess = lower_guess + GUESSES - 1;
	const int log_guesses = 31 - __clz(GUESSES);
	const int option = (blockIdx.x * (parallel_threads_per_block >> log_guesses)) + (tid >> log_guesses);

	/* Build guesses */
	__shared__ float v_ladder[vega_parallel_threads_per_block];
	__shared__ float vega_ladder[vega_parallel_threads_per_block];
	__shared__ float err_ladder[vega_parallel_threads_per_block];
	__shared__ float ferr_ladder[vega_parallel_threads_per_block];

	float ladder_span = 0.04f;	/* Span the guesses over 4% */
	float v_mid = v[option];

	int i = 0;
	int min_err_pos;
	const float guess_fraction = guess / static_cast<float>(GUESSES);
	do
	{
		/* Work out guess */
		v_ladder[tid] = v_mid + (ladder_span * guess_fraction) - (ladder_span * 0.5f) + (ladder_span / static_cast<float>(GUESSES)) * 0.5f;

		/* Price */
		err_ladder[tid] = p[option] - call_price(s, r, v_ladder[tid], t[option], k[option]);
		vega_ladder[tid] = call_vega(s, r, v_ladder[tid], t[option], k[option]);
		ferr_ladder[tid] = abs(err_ladder[tid]);
		
		/* Find minimum error */
		min_err_pos = find_best_guess<GUESSES, vega_parallel_threads_per_block>(ferr_ladder, tid, lower_guess);

		/* Pick the span for the next ladder */
		v_mid = v_ladder[min_err_pos] + (err_ladder[min_err_pos] / vega_ladder[min_err_pos]);
		if ((v_mid < v_ladder[upper_guess]) && (v_mid > v_ladder[lower_guess]))
		{
			ladder_span *= (1.0f / 1024.0f);
		}
		else
		{
			ladder_span *= 2.0f;
		}
	} while ((ferr_ladder[min_err_pos] > tol) && (i++ < iter));

	if (tid == lower_guess)
	{
		v[option] = v_ladder[min_err_pos];
	}
}


void volatility_inversion(const float s, const float r, float *v, const float *t, const float *k, 
	const float *p, const float tol, const int iter, const int num, const kernel_t kernel)
{
	/* Start the profiler */
	//hipProfilerStart();

	/* Get device */
	HANDLE_CUDA_ERROR(hipSetDevice(0));

	/* Allocate and copy memory */
	float *dev_v;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_v, num * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_v, v, num * sizeof(float), hipMemcpyHostToDevice));

	float *dev_t;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_t, num * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_t, t, num * sizeof(float), hipMemcpyHostToDevice));
	
	float *dev_k;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_k, num * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_k, k, num * sizeof(float), hipMemcpyHostToDevice));

	float *dev_p;
	HANDLE_CUDA_ERROR(hipMalloc((void **)&dev_p, num * sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_p, p, num * sizeof(float), hipMemcpyHostToDevice));

	/* Run kernels */
	const int serial_threads = min(num, serial_threads_per_block);
	const int serial_blocks = (int)ceil(num / static_cast<float>(serial_threads));

	const int parallel_threads = parallel_threads_per_block;
	const int parallel_blocks = static_cast<int>(ceil((parallel_guesses_per_option * num) / static_cast<float>(parallel_threads_per_block)));

	const int vega_parallel_threads = vega_parallel_threads_per_block;
	const int vega_parallel_blocks = static_cast<int>(ceil((vega_parallel_guesses_per_option * num) /  static_cast<float>(vega_parallel_threads_per_block)));
	switch (kernel)
	{
		case serial :
			volatility_inversion_device<<<serial_blocks, serial_threads>>>(s, r, dev_v, dev_t, dev_k, dev_p, tol, iter, num);
			break;
		case parallel :
			parallel_volatility_inversion_device<parallel_guesses_per_option><<<parallel_blocks, parallel_threads>>>(s, r, dev_v, dev_t, dev_k, dev_p, tol, iter);
			break;
		case vega_parallel :
			vega_guided_parallel_volatility_inversion_device<vega_parallel_guesses_per_option><<<vega_parallel_blocks, vega_parallel_threads>>>(s, r, dev_v, dev_t, dev_k, dev_p, tol, iter);
			break;
	}

	/* Copy data back */
	HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	HANDLE_CUDA_ERROR(hipMemcpy(v, dev_v, num * sizeof(float), hipMemcpyDeviceToHost));

	/* Clean up */
	HANDLE_CUDA_ERROR(hipFree(dev_v));
	HANDLE_CUDA_ERROR(hipFree(dev_t));
	HANDLE_CUDA_ERROR(hipFree(dev_k));
	HANDLE_CUDA_ERROR(hipFree(dev_p));

	/* Flush profiling info */
	HANDLE_CUDA_ERROR(hipDeviceReset());

	/* Stop the profiler */
	//hipProfilerStop();
}