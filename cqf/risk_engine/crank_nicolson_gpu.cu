
#include <hip/hip_runtime.h>
#include <algorithm>

#include <stdio.h>

#include "hip/hip_runtime_api.h"

/* Scratch space positions */
const int max_ns    = 1024;

const int lower_delta_pos       = 0;
const int mid_delta_pos         = lower_delta_pos + max_ns;
const int upper_delta_pos       = mid_delta_pos + max_ns;

const int lower_gamma_pos       = upper_delta_pos + max_ns;
const int mid_gamma_pos         = lower_gamma_pos + max_ns;
const int upper_gamma_pos       = mid_gamma_pos + max_ns;

const int matrix_equal_pos      = upper_gamma_pos + max_ns;

const int scratch_space_size    = matrix_equal_pos + max_ns;

/* Shared space partition */
const int matrix_lower_pos      = 0;
const int matrix_mid_pos        = matrix_lower_pos + max_ns;
const int matrix_upper_pos      = matrix_mid_pos + max_ns;



void print_cuda_error(hipError_t err, char *at)
{
    if (err)
    {
        printf("Error from CUDA at : %s\n", at);
        printf("Message: %s\n", hipGetErrorString(err));
    }
}


__device__ float call_payoff(const float s, const float k)
{
    return fmaxf(0.0f, s - k);
}


__device__ float asian_call_payoff(const float s, const float a, const float k, const int asianings)
{
    const float val = (a / (asianings - 1)) + (s / asianings);
    return fmaxf(0.0f, val - k);
}


__device__ void get_coeffs(const float *const grid, float *const scratch, const int ns, const int i)
{
    /* Difference vs. the grid below */
    float d0;
    float d1;
    if (i == 0)
    {
        d0 = grid[1] - grid[0];
        d1 = grid[2] - grid[1];
    }
    else if (i == (ns - 1))
    {
        d0 = grid[i - 1] - grid[i - 2];
        d1 = grid[i] - grid[i - 1];
    }
    else
    {
        d0 = grid[i] - grid[i - 1];
        d1 = grid[i + 1] - grid[i];
    }
    const float d1_p_d2 = d0 + d1;

    /* Delta coeffs */
    /* Middle */
    if ((i != 0) & (i != (ns - 1)))
    {
        scratch[lower_delta_pos + i] = -d1 / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = (d1 - d0) / (d0 * d1);
        scratch[upper_delta_pos + i] = d0 / (d1 * d1_p_d2);
    }
    /* Lower boundary */
    else if (i == 0)
    {
        scratch[lower_delta_pos + i] = (-2.0f * d0 - d1) / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = d1_p_d2 / (d0 * d1);
        scratch[upper_delta_pos + i] = -d0 / (d1 * d1_p_d2);
    }
    /* Upper boundary */
    else if (i == (ns - 1))
    {
        scratch[lower_delta_pos + i] = d1 / (d0 * d1_p_d2);
        scratch[mid_delta_pos   + i] = (-d0 - d1) / (d0 * d1);
        scratch[upper_delta_pos + i] = (d0 + 2.0f * d1) / (d1 * d1_p_d2);
    }

    /* Gamma coeffs */
    /* Middle */
    if ((i != 0) & (i != (ns - 1)))
    {
        scratch[lower_gamma_pos + i]  =  2.0f / (d0 * d1_p_d2);
        scratch[mid_gamma_pos   + i]  = -2.0f / (d0 * d1);
        scratch[upper_gamma_pos + i]  =  2.0f / (d1 * d1_p_d2);
    }
    __syncthreads();
}


/* Populate the matrix */
__device__ void populate_matrix(float *const scratch, float *const matrix, float *const matrix_equal, const float *const tp1, const float *const grid, 
    const float half_sigma_sq, const float r, const float t_inc, const int ns, const int i)
{
    /* Boundary conditions */
    /* s = 0.0 */
    if (i == 0)
    {
        const float b = -r * 0.5f * t_inc;
        matrix[matrix_mid_pos  ] = 1.0f - b;
        matrix[matrix_upper_pos] = 0.0f;
        
        matrix_equal[0] = (1.0f + b) * tp1[0];
    }
    /* s = s_max*/
    else if (i == (ns - 1))
    {
        const float r_s = r * grid[ns - 1];

        const float a = -r_s * 0.5f * t_inc;
        const float b = -(r - r_s) * 0.5f * t_inc;
        
        matrix[matrix_lower_pos + ns - 1] = -a;
        matrix[matrix_mid_pos   + ns - 1] = 1.0f - b;
        
        matrix_equal[ns - 1]  = a * tp1[ns - 2];
        matrix_equal[ns - 1] += (1.0f + b) * tp1[ns - 1];
    }
    else if (i < ns)
    {
        const float g = half_sigma_sq * grid[i] * grid[i];
        const float r_s = r * grid[i];
            
        const float a = ((scratch[lower_delta_pos + i] * r_s) + (scratch[lower_gamma_pos + i] * g))     * 0.5f * t_inc;
        const float b = ((scratch[mid_delta_pos + i]   * r_s) + (scratch[mid_gamma_pos + i]   * g) - r) * 0.5f * t_inc;
        const float c = ((scratch[upper_delta_pos + i] * r_s) + (scratch[upper_gamma_pos + i] * g))     * 0.5f * t_inc;
            
        matrix[matrix_lower_pos + i] = -a;
        matrix[matrix_mid_pos   + i] = 1.0f - b;
        matrix[matrix_upper_pos + i] = -c;
            
        matrix_equal[i]  = a * tp1[i - 1];
        matrix_equal[i] += (1.0f + b) * tp1[i];
        matrix_equal[i] += c * tp1[i + 1];
    }

    __syncthreads();
}


__device__ int cyclic_reduction_forward_reduction(float *lower, float *diagonal, float *upper, float *equal, const int dim, int step, const int to)
{
    /* Forward reduction */
    for (; (step * to * 3) <= dim; step <<= 1)
    {
        const int addr = (threadIdx.x * (step << 1)) + (step << 1) - 1;
        if (addr < dim)
        {
            if (addr - step >= 0)
            {
                const float alpha = -lower[addr] / diagonal[addr - step];
                equal[addr]    += (alpha * equal[addr - step]);
                diagonal[addr] += (alpha * upper[addr - step]);
                lower[addr]     = alpha * lower[addr - step];
            }

            if (addr + step < dim)
            {
                const float gamma = -upper[addr] / diagonal[addr + step];
                equal[addr]    += (gamma * equal[addr + step]);
                diagonal[addr] += (gamma * lower[addr + step]);
                upper[addr]     = gamma * upper[addr + step];
            }
        }
        __syncthreads();
    }

    return step;
}


__device__ void cyclic_reduction_back_substitution(float *lower, float *diagonal, float *upper, float *equal, const int dim, int step, const int to)
{
    /* Backward substitution */
    for (; step > to; step >>= 1)
    {
        const int addr = (threadIdx.x * (step << 1)) + step - 1;
        if (addr < dim)
        {
            if (addr - step >= 0)
            {
                equal[addr] -= (lower[addr] * equal[addr - step]);
            }

            if (addr + step < dim)
            {
                equal[addr] -= (upper[addr] * equal[addr + step]);
            }

            equal[addr] = equal[addr] / diagonal[addr];
        }
        __syncthreads();
    }
}


__device__ void cyclic_reduction_device(float *lower, float *diagonal, float *upper, float *equal, const int dim)
{
    /* Forward reduction */
    int step = cyclic_reduction_forward_reduction(lower, diagonal, upper, equal, dim, 1, 1);

    /* Solve base system */
    if (threadIdx.x == 0)
    {
        if ((dim / step) == 2) /* Solve simultaneous equations */
        {
            const int equal_addr = (step << 1) - 1;
            const float a0 = diagonal[equal_addr - step];
            const float a1 = lower[equal_addr];
            const float b0 = upper[equal_addr - step];
            const float b1 = diagonal[equal_addr];
            const float c0 = equal[equal_addr - step];
            const float c1 = equal[equal_addr];

            equal[equal_addr] = (c0 * a1 - a0 * c1) / (a1 * b0 - a0 * b1);
            equal[equal_addr - step] = (c0 - b0 * equal[equal_addr]) / a0;
        }
        else /* blk_size == 1, equations are already solved */
        {
            const int equal_addr = step - 1;
            equal[equal_addr] = equal[equal_addr] / diagonal[equal_addr];
        }
    }

    __syncthreads();
    step >>= 1;

    /* Backward substitution */
    cyclic_reduction_back_substitution(lower, diagonal, upper, equal, dim, step, 0);
}


__device__ void parallel_cyclic_reduction(float *l, float *d, float *u, float *h, const int ns, const int i)
{
    for (int step = 1; step < ns; step <<= 1)
    {
        float h_tmp = h[i];
        float d_tmp = d[i];

        float l_tmp;
        if (i - step >= 0)
        {
            l_tmp = -l[i] / d[i - step ];

            d_tmp += l_tmp * u[i - step];
            h_tmp += l_tmp * h[i - step];
            l_tmp *= l[i - step];
        }

        float u_tmp;
        if (i + step < ns)
        {
            u_tmp = -u[i] / d[i + step];

            d_tmp += u_tmp * l[i + step];
            h_tmp += u_tmp * h[i + step];
            u_tmp *= u[i + step];
        }
        __syncthreads();

        l[i] = l_tmp;
        u[i] = u_tmp;
        h[i] = h_tmp;
        d[i] = d_tmp;
        __syncthreads();
    }

    h[i] /= d[i];
    __syncthreads();
}


__device__ void solve_tridiagonal(float *const matrix, float *const matrix_equal, const int ns, const int i)
{
    //cyclic_reduction_device(&matrix[matrix_lower_pos], &matrix[matrix_mid_pos], &matrix[matrix_upper_pos], matrix_equal, ns);
    parallel_cyclic_reduction(&matrix[matrix_lower_pos], &matrix[matrix_mid_pos], &matrix[matrix_upper_pos], matrix_equal, ns, i);
}


__global__ void crank_nicolson(const float *const grid, float *const scratch, const float half_sigma_sq, const float r, 
    const float t_inc, const float k, const int ns, const int nt)
{
    const int i = threadIdx.x;
    if (ns & 0x1f)
    {
        /* Only multiple of 32 space steps are supported */
        return;
    }
    
    /* Move grid to shared memory, needed for off by 1 access and reused */
    __shared__ float shared_equal[max_ns];
    shared_equal[i] = grid[i];
    __syncthreads();

    /* Build grid based coeffs, completely parrallel */
    __shared__ float shared_tp1[max_ns];
    shared_tp1[i] = call_payoff(shared_equal[i], k);
    get_coeffs(shared_equal, scratch, ns, i);
    
    /* Solve back in time */
    __shared__ float shared_matrix[3 * max_ns];
    for (unsigned int j = 0; j < nt >> 1; ++j)
    {
        populate_matrix(scratch, shared_matrix, shared_equal, shared_tp1, grid, half_sigma_sq, r, t_inc, ns, i);
        solve_tridiagonal(shared_matrix, shared_equal, ns, i);
        shared_equal[i] = fmaxf(shared_equal[i], call_payoff(shared_equal[i], k));
        __syncthreads();

        populate_matrix(scratch, shared_matrix, shared_tp1, shared_equal, grid, half_sigma_sq, r, t_inc, ns, i);
        solve_tridiagonal(shared_matrix, shared_tp1, ns, i);
        shared_tp1[i] = fmaxf(shared_tp1[i], call_payoff(shared_tp1[i], k));
        __syncthreads();
    }

    scratch[matrix_equal_pos + i] = shared_tp1[i];
}


__global__  void transpose(float *const trans, const int x, const int y)
{
    /* Position of this thread in a tranpose block */
    const int blk_size = 32;
    const int x_offset = threadIdx.x >> 5;
    const int y_offset = threadIdx.x & (blk_size - 1);

    const int x_blks = x & ~(blk_size - 1);
    const int y_blks = y & ~(blk_size - 1);

    __shared__ float shared_trans[blk_size][blk_size + 1][2]; /* 32x32 block of transposed data */

    /* Transpose off diagonal blocks */
    for (int i = blk_size; i < x_blks; i += blk_size)
    {
        for (int j = 0; j < i; j += blk_size)
        {
            /* Streaming load and within block transposed save into shared */
            shared_trans[y_offset][x_offset][0] = trans[((i + x_offset) * y) + j + y_offset];
            shared_trans[y_offset][x_offset][1] = trans[((j + x_offset) * y) + i + y_offset];

            __syncthreads();

            /* Streaming save */
            trans[((i + x_offset) * y) + j + y_offset] = shared_trans[x_offset][y_offset][1];
            trans[((j + x_offset) * y) + i + y_offset] = shared_trans[x_offset][y_offset][0];
        }
    }

    /* Transpose diagonal blocks */
    for (int i = 0; i < min(x_blks, y_blks); i += blk_size)
    {
        /* Streaming load and within block transposed save into shared */
        const int blk_addr = ((i + x_offset) * y) + i + y_offset;
        shared_trans[y_offset][x_offset][0] = trans[blk_addr];

        __syncthreads();

        /* Streaming save */
        trans[blk_addr] = shared_trans[x_offset][y_offset][0];
    }
}


void transpose_test()
{
    const int x = 1024;
    const int y = 1024;
    float *trans = new float [x * y];
    for (int i = 0; i < x * y; ++i)
    {
        trans[i] = i;
    }

    print_cuda_error(hipSetDevice(0), "Set device");

    /* Prepare device memory */
    float *dev_trans;
    print_cuda_error(hipMalloc((void **)&dev_trans, x * y * sizeof(float)), "Malloc matrix");
    print_cuda_error(hipMemcpy(dev_trans, trans, x * y * sizeof(float), hipMemcpyHostToDevice), "Copy martix to device");

    /* Call kernel */
    hipProfilerStart();
    transpose<<<1, 1024>>>(dev_trans, x, y);
    hipProfilerStop();
    print_cuda_error(hipGetLastError(), "Kernel execution");

    print_cuda_error(hipMemcpy(trans, dev_trans, x * y * sizeof(float), hipMemcpyDeviceToHost), "Copy matrix to host");
    // for (int i = 0; i < x; ++i)
    // {
    //     for (int j = 0; j < y; ++j)
    //     {
    //         printf("%.2f ", trans[(i * y) + j]);
    //     }
    //     printf("\n");
    // }

    /* Clean up */
    print_cuda_error(hipFree(dev_trans), "Free matrix");
    
    print_cuda_error(hipDeviceReset(), "Device reset");

    delete [] trans;
}


__device__ __host__ float interpolate(const float *const x, const float *const y, const float a, const int ns, const int a_idx, const int s_idx)
{
    /* Linear interpolation */
    const int y_idx = (a_idx * ns) + s_idx;
    return y[y_idx - ns] + ((y[y_idx] - y[y_idx - ns]) * ((a - x[a_idx - 1]) / (x[a_idx] - x[a_idx - 1])));
}


// __global__ void permute_asian_values(float *const v_grid1, const float *const a_grid1, const float *const a_grid0, const float *const s_grid, 
//     const float *const v_grid0, const float a_fac, const float s_fac, const float a_inc_inv, const int ns, const int na0, const int na1)
// {
//     const int y_lower = blockIdx.y * blockDim.x;
//     const int y_upper = (blockIdx.y * blockDim.x) + blockDim.x;

//     const int i = (blockIdx.x * blockDim.x) + threadIdx.x;

//     int cache_idx = threadIdx.x;
//     __shared__ float cache[64 * 64];
//     for (int j = y_lower; j < y_upper; ++j)
//     {
//         cache[cache_idx] = v_grid0[(j * ns) + i];
//         cache_idx += blockDim.x;
//     }

//     /* I would fix at s_value */
//     const float s_value = s_grid[i];

//     int a1_idx = 0;
//     const float prefix_asian_value = a_grid1[a1_idx];

//     /* After fixing I was at postfix_asian_value */
//     const float postfix_asian_value = (prefix_asian_value * a_fac) + (s_value * s_fac);
    
//     /* Find and interpolate around the postfix_asian_value */
//     int a0_idx = max(y_lower + 1, min(y_upper, static_cast<int>(postfix_asian_value * a_inc_inv)));
//     for (int j = y_lower; j < y_upper; ++j)
//     {
//         while (a0_idx == j)
//         {
//             v_grid1[(a1_idx * ns) + i] = interpolate(a_grid0, cache, postfix_asian_value, blockDim.x, a0_idx - y_lower, threadIdx.x);

//             const float prefix_asian_value = a_grid1[++a1_idx];
//             const float postfix_asian_value = (prefix_asian_value * a_fac) + (s_value * s_fac);
//             a0_idx = max(y_lower + 1, min(y_upper, static_cast<int>(postfix_asian_value * a_inc_inv)));
//         }
//     }
// }


__global__ void permute_asian_values(float *const v_grid1, const float *const a_grid1, const float *const a_grid0, const float *const s_grid, 
    const float *const v_grid0, const float a_fac, const float s_fac, const float a_inc_inv, const int ns, const int na0, const int na1)
{
    const int i = threadIdx.x;

    /* I would fix at s_value */
    const float s_value = s_grid[i];

    int a1_idx = 0;
    const float prefix_asian_value = a_grid1[a1_idx];

    /* After fixing I was at postfix_asian_value */
    const float postfix_asian_value = (prefix_asian_value * a_fac) + (s_value * s_fac);
    
    /* Find and interpolate around the postfix_asian_value */
    int a0_idx = max(1, min(na0, static_cast<int>(postfix_asian_value * a_inc_inv)));
    for (int j = 0; j < na0; ++j)
    {
        while (a0_idx == j)
        {
            v_grid1[(a1_idx * ns) + i] = interpolate(a_grid0, v_grid0, postfix_asian_value, ns, a0_idx, i);

            const float prefix_asian_value = a_grid1[++a1_idx];
            const float postfix_asian_value = (prefix_asian_value * a_fac) + (s_value * s_fac);
            a0_idx = max(1, min(na0, static_cast<int>(postfix_asian_value * a_inc_inv)));
        }
    }
}


__host__ int update_asian_grid(const float *const grid_ping, float *const grid_pong, const int na, const int t_idx)
{
    const int t_idx_m1          = t_idx - 1;
    const float flt_t_idx       = static_cast<float>(t_idx);
    const float t_idx_inv       = 1.0f / flt_t_idx;
    const float t_idx_m1_inv    = 1.0f / (flt_t_idx - 1.0f);
    const float a_fac           = (flt_t_idx - 1.0f) * t_idx_inv;

    /* Asianing update */
    /* Rebuild uniform asian grid */
    /* This may be slightly too big, but not too much to worry about */
    /* This loop will get all, but the last t_idx - 1 or less points */
    const unsigned int whole_iters = (na - 1) / t_idx;
    for (unsigned int i = 0; i < whole_iters; i++)
    {
        const unsigned int l_idx = i * t_idx;
        const unsigned int h_idx = l_idx + t_idx;
        const float step = (grid_ping[h_idx] - grid_ping[l_idx]) * t_idx_m1_inv;
        for (unsigned int j = 0; j < t_idx_m1; j++)
        {
            grid_pong[(i * t_idx_m1) + j] = grid_ping[l_idx] + (step * static_cast<float>(j));
        }
    }
    
    /* This loop will get the remaining points */
    const unsigned int part_iters = na - (whole_iters * t_idx);
    const unsigned int l_idx = na - part_iters - 1;
    const float step = (grid_ping[na - 1] - grid_ping[l_idx]) * t_idx_m1_inv;
    for (unsigned int i = 0; i < part_iters; i++)
    {
        grid_pong[(whole_iters * t_idx_m1) + i] = grid_ping[l_idx] + (step * static_cast<float>(i));
    }
    grid_pong[(whole_iters * t_idx_m1) + part_iters - 1] = grid_ping[na - 1];

    return (na * a_fac) + 1;
}


__global__ void crank_nicolson_asian(const float *const grid, float *const glb_values, float *const glb_scratch, const float half_sigma_sq, 
    const float r, const float t_inc, const float k, const int ns, const int nt, const int asianings, const bool create_payoff)
{
    const int i = threadIdx.x;
    const int asian_idx = blockIdx.x;
    float *const scratch = &glb_scratch[asian_idx * scratch_space_size];
    float *const values = &glb_values[asian_idx * ns];
    if (ns & 0x1f)
    {
        /* Only multiple of 32 space steps are supported */
        return;
    }
    
    /* Move grid to shared memory, needed for off by 1 access and reused */
    __shared__ float shared_equal[max_ns];
    shared_equal[i] = grid[i];
    __syncthreads();

    /* Build grid based coeffs, completely parrallel */
    __shared__ float shared_tp1[max_ns];
    if (create_payoff)
    {
        shared_tp1[i] = asian_call_payoff(shared_equal[i], grid[ns + asian_idx], k, asianings);
    }
    else
    {
        shared_tp1[i] = values[i];
    }
    get_coeffs(shared_equal, scratch, ns, i);
    
    /* Solve back in time */
    __shared__ float shared_matrix[3 * max_ns];
    for (unsigned int j = 0; j < nt >> 1; ++j)
    {
        populate_matrix(scratch, shared_matrix, shared_equal, shared_tp1, grid, half_sigma_sq, r, t_inc, ns, i);
        solve_tridiagonal(shared_matrix, shared_equal, ns, i);
        shared_equal[i] = fmaxf(shared_equal[i], call_payoff(shared_equal[i], k));
        __syncthreads();

        populate_matrix(scratch, shared_matrix, shared_tp1, shared_equal, grid, half_sigma_sq, r, t_inc, ns, i);
        solve_tridiagonal(shared_matrix, shared_tp1, ns, i);
        shared_tp1[i] = fmaxf(shared_tp1[i], call_payoff(shared_tp1[i], k));
        __syncthreads();
    }

    values[i] = shared_tp1[i];
}


void asian_call_test()
{
    /* Pricing set up */
    printf("Pricing Asian Call\n");
    const unsigned int ns = 1024;   /* Want multiples of warp size (32) */

    const int asianings = 2;
    const float t_inc = 0.01;
    const float t[asianings] = { 0.9f, 1.0f };
    const float k = 100.0f;

    const float s = 100.0f;
    const float r = 0.05f;
    const float sigma = 0.2f;
    const float half_sigma_sq = 0.5f * sigma * sigma;

    /* Build regular grid based at 0 */
    int na = 1024;
    const int grid_size = ns + na;
    float *grid_ping = new float [grid_size];

    const float s_inc = (s * 3.0f) / ns;
    for (int i = 0; i < ns; ++i)
    {
        grid_ping[i] = i * s_inc;
    }

    const float a_inc = (s * 3.0f) / na;
    for (int i = 0; i < na; ++i)
    {
        grid_ping[ns + i] = i * a_inc;
    }

    float *grid_pong = new float [grid_size];
    memcpy(grid_pong, grid_ping, ns * sizeof(float));


    /* Set cuda device */
    print_cuda_error(hipSetDevice(0), "Set device");

    /* Prepare device memory */
    float *dev_grid;
    print_cuda_error(hipMalloc((void **)&dev_grid, 2 * grid_size * sizeof(float)), "Malloc grid");
    print_cuda_error(hipMemcpy(dev_grid, grid_ping, grid_size * sizeof(float), hipMemcpyHostToDevice), "Copy grid to device");
    float *dev_grid_ping = &dev_grid[0];
    float *dev_grid_pong = &dev_grid[grid_size];

    float *values = new float [ns * na];

    float *dev_values;
    print_cuda_error(hipMalloc((void **)&dev_values, 2 * ns * na * sizeof(float)), "Malloc values");
    print_cuda_error(hipMemset(dev_values, 0, ns * na * sizeof(float)), "Clear values");
    float *dev_values_ping = &dev_values[0];
    float *dev_values_pong = &dev_values[ns * na];

    float *dev_scratch;
    print_cuda_error(hipMalloc((void **)&dev_scratch, na * scratch_space_size * sizeof(float)), "Malloc scratch");
    print_cuda_error(hipMemset(dev_scratch, 0, na * scratch_space_size * sizeof(float)), "Clear scratch");

    /* Call kernel and update asian grid in parallel */
    hipProfilerStart();

    /* Work back through the asianings */
    for (int i = asianings; i > 1; --i)
    {
        /* Time step the asian slices */
        const int nt = (t[asianings - 1] - t[asianings - 2]) / t_inc;
        crank_nicolson_asian<<<na, ns>>>(dev_grid_ping, dev_values_ping, dev_scratch, half_sigma_sq, r, t_inc, k, ns, nt, i, (i == asianings));
        
        /* Update the asian grid in parallel */
        const float flt_t_idx   = static_cast<float>(i);
        const float t_idx_inv   = 1.0f / flt_t_idx;
        const float a_fac       = (flt_t_idx - 1.0f) * t_idx_inv;
        const int na_lst        = na;
        na = update_asian_grid(&grid_ping[ns], &grid_pong[ns], na, i);
        std::swap(grid_ping, grid_pong);
        std::swap(dev_grid_ping, dev_grid_pong);
        std::swap(dev_values_ping, dev_values_pong);
        print_cuda_error(hipMemcpy(dev_grid_ping, grid_ping, grid_size * sizeof(float), hipMemcpyHostToDevice), "Copy grid to device");

        /* Wait for the device to complete */
        print_cuda_error(hipDeviceSynchronize(), "Synchronise device");
        // print_cuda_error(cudaMemcpy(values, dev_values_pong, ns * na_lst * sizeof(float), cudaMemcpyDeviceToHost), "Copy values to host");
        // printf("      ");
        // for (int i = ns; i < ns + na_lst; ++i)
        // {
        //     printf("%.2f ", grid_pong[i]);
        // }
        // printf("\n");

        // for (unsigned int i = 0; i < ns; ++i)
        // {
        //     printf("%.2f: ", grid_ping[i]);
        //     for (int j = 0; j < na_lst; ++j)
        //     {
        //         printf("%.2f ", values[(j * ns)+ i]);
        //     }
        //     printf("\n");
        // }
        // printf("\n\n");

        /* Update the grid values for asianing */
        dim3 grid(ns >> 6, na >> 6);
        const int a_inc_inv = na_lst / (s * 3.0f);
        permute_asian_values<<<1, ns>>>(dev_values_ping, &dev_grid_ping[ns], &dev_grid_pong[ns], dev_grid_pong, dev_values_pong, 
            a_fac, t_idx_inv, a_inc_inv, ns, na_lst, na);

        /* Wait for the device to complete */
        print_cuda_error(hipDeviceSynchronize(), "Synchronise device");
        // print_cuda_error(cudaMemcpy(values, dev_values_ping, ns * na * sizeof(float), cudaMemcpyDeviceToHost), "Copy values to host");
        // printf("      ");
        // for (int i = ns; i < ns + na; ++i)
        // {
        //     printf("%.2f ", grid_ping[i]);
        // }
        // printf("\n");

        // for (unsigned int i = 0; i < ns; ++i)
        // {
        //     printf("%.2f: ", grid_ping[i]);
        //     for (int j = 0; j < na; ++j)
        //     {
        //         printf("%.2f ", values[(j * ns)+ i]);
        //     }
        //     printf("\n");
        // }
        // printf("\n\n");
    }

    /* Final period */
    const int nt = t[0] / t_inc;
    crank_nicolson_asian<<<1, ns>>>(dev_grid_ping, dev_values_ping, dev_scratch, half_sigma_sq, r, t_inc, k, ns, nt, 0, false);

    hipProfilerStop();
    print_cuda_error(hipGetLastError(), "Kernel execution");


    float *res = new float [ns];
    print_cuda_error(hipMemcpy(res, &dev_values_ping[0], ns * sizeof(float), hipMemcpyDeviceToHost), "Copy grid to host");
    for (unsigned int i = 0; i < ns; ++i)
    {
        printf("%.2f: %.2f\n", grid_ping[i], res[i]);
    }

    /* Clean up */
    print_cuda_error(hipFree(dev_grid), "Free grid");
    print_cuda_error(hipFree(dev_scratch), "Free scratch");

    print_cuda_error(hipDeviceReset(), "Device reset");

    delete [] grid_ping;
    delete [] grid_pong;
    delete [] res;
    delete [] values;
}


void american_call_test()
{
    /* Pricing set up */
    printf("Pricing American Call\n");
    const unsigned int ns = 1024;   /* Want multiples of warp size (32) */
    const unsigned int nt = 100;

    const float k = 100.0f;
    const float t = 1.0f;
    const float t_inc = t / nt;

    const float s = 100.0f;
    const float r = 0.05f;
    const float sigma = 0.2f;
    const float half_sigma_sq = 0.5f * sigma * sigma;

    /* Build regular grid based at 0 */
    float *grid = new float [ns];
    const float s_inc = (s * 3.0f) / ns;
    for (unsigned int i = 0; i < ns; ++i)
    {
        grid[i] = i * s_inc;
    }

    print_cuda_error(hipSetDevice(0), "Set device");

    /* Prepare device memory */
    float *dev_grid;
    print_cuda_error(hipMalloc((void **)&dev_grid, ns * sizeof(float)), "Malloc grid");
    print_cuda_error(hipMemcpy(dev_grid, grid, ns * sizeof(float), hipMemcpyHostToDevice), "Copy grid to device");

    float *dev_scratch;
    print_cuda_error(hipMalloc((void **)&dev_scratch, scratch_space_size * sizeof(float)), "Malloc scratch");
    print_cuda_error(hipMemset(dev_scratch, 0, scratch_space_size * sizeof(float)), "Clear scratch");

    /* Call kernels */
    hipProfilerStart();
    crank_nicolson<<<1, ns>>>(dev_grid, dev_scratch, half_sigma_sq, r, t_inc, k, ns, nt);
    hipProfilerStop();
    print_cuda_error(hipGetLastError(), "Kernel execution");

    float *res = new float [ns];
    print_cuda_error(hipMemcpy(res, &dev_scratch[matrix_equal_pos], ns * sizeof(float), hipMemcpyDeviceToHost), "Copy grid to host");
    for (unsigned int i = 0; i < ns; ++i)
    {
        printf("%.2f: %.2f\n", grid[i], res[i]);
    }

    /* Clean up */
    print_cuda_error(hipFree(dev_grid), "Free grid");
    print_cuda_error(hipFree(dev_scratch), "Free scratch");

    print_cuda_error(hipDeviceReset(), "Device reset");

    delete [] grid;
    delete [] res;
}


int main()
{
    american_call_test();
    //asian_call_test();
    //transpose_test();

    return 0;
}


/* Return the first index not less than a */
__device__ __host__ int search(const float *const x, const float a, int i, const int s)
{
    if (x[i] < a)
    {
        while ((i < (s - 1)) && (x[i] < a))
        {
            ++i;
        }
    }
    /* Values are correlates and down, begin linear search downwards */
    else
    {
        while ((i > 1) && (x[i - 1] >= a))
        {
            --i;
        }
    }
    
    return i;
}
